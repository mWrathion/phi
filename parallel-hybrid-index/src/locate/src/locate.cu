#include "hip/hip_runtime.h"
#include "../../kernels.cuh"

__device__ u_long getNum64(u_long* A, u_int ini, u_int len){
    u_long i=ini>>6, j=ini-(i<<6);
    u_long result = (A[i] << j) >> (64-len);

    if (j+len > 64)
        result = result | (A[i+1] >> (128-j-len));

    return result;
}

__device__ bool isPrimary(u_long x, u_int len, u_long *pIni, u_int *dIni, 
                          u_int nSamP, u_int lgN, u_int POT_GC, u_long* SGCPFT, 
                          u_int lgPFT, u_long* PhraFT){
	u_long l=0, r=nSamP-1, m=nSamP>>1, x1, x2, phr;

	while (l<=r){
		x1 = getNum64(SGCPFT, m*lgN, lgN);
		if (x1 <= x){
			if (m<r){
				x2 = getNum64(SGCPFT, (m+1)*lgN, lgN);
				if (x2 > x){
					phr = m<<POT_GC;
					break;
				}else{
					l=m+1;
					m=(l+r)>>1;
				}
			}else{
				phr = m<<POT_GC;
				break;
			}
		}else {
			m--;
			if (l<=m){
				x1 = getNum64(SGCPFT, m*lgN, lgN);
				if (x1 <= x){
					phr = m<<POT_GC;
					break;
				}else{
					r=m-1;
					m=(l+r)>>1;
				}
			}else {
				phr = m<<POT_GC;
				break;
			}
		}
	}

	// retrieve gaps from x1 to x
	for(r = (phr+1)*lgPFT; x1<x; phr++, r+=lgPFT)
		x1 += getNum64(PhraFT, r, lgPFT);

	if (x1 == x){
		*dIni = 0;
		*pIni = phr;
		x1 += getNum64(PhraFT, r, lgPFT);
	}else {
		*pIni = phr;
		*dIni = x1-x;
	}

	if (x+len <= x1)
		return false;

	return true;
}

__global__ void getPrimaryOccurrences(u_char* text, u_int size, u_char* pattern, u_int m, int *nOcc, 
                                      bool* BL_il,
                                      u_int nSamP, u_int lgN, u_int POT_GC, u_long* SGCPFT, 
                                      u_int lgPFT, u_long* PhraFT){
    u_int tid = blockIdx.x * blockDim.x + threadIdx.x;
	bool flag = true;
	u_int c = 0;
	
	if (tid < size - m){
		for (int i = tid; i < tid+m; ++i){
			if(text[i] != pattern[c]){	
				flag = false;	
			}
			c++;
		}
		if (flag == 1) {  
			long id = atomicAdd(nOcc, 1);
 
           /* u_long pr;
            u_int dx;
            long id = -1;
            if(isPrimary(tid, m, &pr, &dx,nSamP, lgN, POT_GC, SGCPFT, lgPFT, PhraFT)){
                
                id = atomicAdd(nOcc, 1);
            }
            else{
                if(dx){
                    if(BL_il[pr-1]){
                        id = atomicAdd(nOcc, 1);
                    }
                }
                else{
                    if(BL_il[pr]){
                        id = atomicAdd(nOcc, 1);
                    }
                }
            }*/
            //occs[tid] = 0; 
        }
    }
}

void locatePrimaryOccurrences(u_char* text, u_int size, u_char* pattern, u_int m, int &nOcc, 
                              bool* BL_il,
                              u_int nSamP, u_int lgN, u_int POT_GC, u_long* SGCPFT, 
                              u_int lgPFT, u_long* PhraFT){
    int *d_cont;
    hipMalloc((void **) &d_cont, sizeof(int));
    hipMemcpy(d_cont, &nOcc, sizeof(int), hipMemcpyHostToDevice);


    getPrimaryOccurrences<<<(size+1023)/1024, 1024>>>(text, size, pattern, m, d_cont, BL_il,
    nSamP, lgN, POT_GC, SGCPFT, lgPFT, PhraFT);

    hipMemcpy(&nOcc, d_cont, sizeof(int), hipMemcpyDeviceToHost);
}