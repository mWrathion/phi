#include "hip/hip_runtime.h"
#include "../../kernels.cuh"

__device__ u_long getNum64(u_long* A, u_int ini, u_int len){
    u_long i=ini>>6, j=ini-(i<<6);
    u_long result = (A[i] << j) >> (64-len);

    if (j+len > 64)
        result = result | (A[i+1] >> (128-j-len));

    return result;
}

__device__ u_long getPosPhraT(ulong phra, u_int lgN, u_int POT_GC, u_long* SGCPT, u_int lgPT, u_long* PhraT){
	ulong ph = phra>>POT_GC;
	ulong x = getNum64(SGCPT, ph*lgN, lgN);
	if (phra%16 == 0)
		return x;

	// extract from the sampled phrase 'ph' to 'phra'...
	ph=(ph<<POT_GC)+1;
	for (ulong c = ph*lgPT; ph<=phra; ph++, c+=lgPT)
		x += getNum64(PhraT, c, lgPT);

	return x;
}

__device__ bool isPrimary(u_long x, u_int len, u_long *pIni, u_int *dIni, 
                          u_int nSamP, u_int lgN, u_int POT_GC, u_long* SGCPFT, 
                          u_int lgPFT, u_long* PhraFT){
	u_long l=0, r=nSamP-1, m=nSamP>>1, x1, x2, phr;

	while (l<=r){
		x1 = getNum64(SGCPFT, m*lgN, lgN);
		if (x1 <= x){
			if (m<r){
				x2 = getNum64(SGCPFT, (m+1)*lgN, lgN);
				if (x2 > x){
					phr = m<<POT_GC;
					break;
				}else{
					l=m+1;
					m=(l+r)>>1;
				}
			}else{
				phr = m<<POT_GC;
				break;
			}
		}else {
			m--;
			if (l<=m){
				x1 = getNum64(SGCPFT, m*lgN, lgN);
				if (x1 <= x){
					phr = m<<POT_GC;
					break;
				}else{
					r=m-1;
					m=(l+r)>>1;
				}
			}else {
				phr = m<<POT_GC;
				break;
			}
		}
	}

	// retrieve gaps from x1 to x
	for(r = (phr+1)*lgPFT; x1<x; phr++, r+=lgPFT)
		x1 += getNum64(PhraFT, r, lgPFT);

	if (x1 == x){
		*dIni = 0;
		*pIni = phr;
		x1 += getNum64(PhraFT, r, lgPFT);
	}else {
		*pIni = phr;
		*dIni = x1-x;
	}

	if (x+len <= x1)
		return false;

	return true;
}

__global__ void getPrimaryOccurrences(u_char* text, u_int size, u_char* pattern, u_int m, int *nOcc, 
                                      bool* BL_il,
                                      u_int nSamP, u_int lgN, u_int POT_GC, u_long* SGCPFT, 
                                      u_int lgPFT, u_long* PhraFT,
									  u_long* SGCPT, u_int lgPT, u_long* PhraT, u_long* occs){
    u_int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= size - m) return;
	
	bool flag = true;
	u_int c = 0;

	for (int i = tid; i < tid+m; ++i){
		if(text[i] != pattern[c]){	
			flag = false;	
		}
		c++;
	}
	if (!flag) return;
	
	u_long pr;
	u_int dx;
	long id;
	if(isPrimary(tid, m, &pr, &dx,nSamP, lgN, POT_GC, SGCPFT, lgPFT, PhraFT)){
		id = atomicAdd(nOcc, 1);
		occs[id] = getPosPhraT(pr, lgN, POT_GC, SGCPT, lgPT, PhraT) - dx;
	}
	else{
		if(dx){
			if(BL_il[pr-1]){
				id = atomicAdd(nOcc, 1);
				occs[id] = getPosPhraT(pr, lgN, POT_GC, SGCPT, lgPT, PhraT) - dx;
			}
		}
		else{
			if(BL_il[pr]){
				id = atomicAdd(nOcc, 1);
				occs[id] = getPosPhraT(pr, lgN, POT_GC, SGCPT, lgPT, PhraT);
			}
		}
	}
}

void locatePrimaryOccurrences(u_char* text, u_int size, u_char* pattern, u_int m, int &nOcc, 
                              bool* BL_il,
                              u_int nSamP, u_int lgN, u_int POT_GC, u_long* SGCPFT, 
                              u_int lgPFT, u_long* PhraFT,
							  u_long* SGCPT, u_int lgPT, u_long* PhraT, u_long* occs, u_long *h_occs){
    int *d_cont;
    hipMalloc((void **) &d_cont, sizeof(int));
    hipMemcpy(d_cont, &nOcc, sizeof(int), hipMemcpyHostToDevice);


    getPrimaryOccurrences<<<(size+1023)/1024, 1024>>>(text, size, pattern, m, d_cont, BL_il,
    							nSamP, lgN, POT_GC, SGCPFT, lgPFT, PhraFT, SGCPT, lgPT, PhraT, occs);

    hipMemcpy(&nOcc, d_cont, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(h_occs, occs, nOcc*sizeof(u_long), hipMemcpyDeviceToHost);
}