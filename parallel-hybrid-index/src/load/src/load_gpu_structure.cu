#include "../../kernels.cuh"

/**
 * @brief Preload every needed array to solve PM into the GPU
 * 
 * @param text The LZ77 filtered text
 * @param size size of the LZ77 filtered text
 * @param patterns Array of patterns to search
 * @param m size of the patterns
 * @param nPatt number of patterns to search
 * @param bitVector bit vector to search literal phrases
 * @param z lenght of the bit vector
 * @return d_data 
 */
d_data loadGPUStructures(u_char *text, u_int size, 
                         u_char** patterns, u_int m, u_int nPatt, 
                         bool* bitVector, u_int z,
                         u_long* SGCPFT, u_int nSamP, u_long* PhraFT){
	u_char **d_pattern = (u_char **) malloc (nPatt*sizeof(u_char*));
	for (int i = 0; i < nPatt; ++i){
		hipMalloc((void **) &d_pattern[i], (m+1)*sizeof(u_char));
		hipMemcpy(d_pattern[i], patterns[i],(m+1)*sizeof(u_char), hipMemcpyHostToDevice);
	}

    int bytes = size*sizeof(u_char);
    u_char* d_text;
	hipMalloc((void **) &d_text, bytes);
	hipMemcpy(d_text, text, bytes, hipMemcpyHostToDevice);

	
	bool* d_bitVector;
	hipMalloc((void**) &d_bitVector,z*sizeof(bool));
	hipMemcpy(d_bitVector, bitVector, z*sizeof(bool), hipMemcpyHostToDevice);

	u_long* d_sgcpft;
	hipMalloc((void**) &d_sgcpft, nSamP*sizeof(ulong));
	hipMemcpy(d_sgcpft, SGCPFT, nSamP*sizeof(ulong), hipMemcpyHostToDevice);

	u_long* d_phraft;
	hipMalloc((void**) &d_phraft, z*sizeof(ulong));
	hipMemcpy(d_phraft, PhraFT, z*sizeof(ulong), hipMemcpyHostToDevice);

	printf("Data structures loaded into the GPU!\n");

	return d_data{
		d_text,
		d_pattern,
		d_bitVector,
		d_sgcpft,
		d_phraft
	};

}