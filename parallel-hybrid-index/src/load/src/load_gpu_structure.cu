#include "../../kernels.cuh"

/**
 * @brief Preload every needed array to solve PM into the GPU
 * 
 * @param text The LZ77 filtered text
 * @param size size of the LZ77 filtered text
 * @param patterns Array of patterns to search
 * @param m size of the patterns
 * @param nPatt number of patterns to search
 * @param bitVector bit vector to search literal phrases
 * @param z lenght of the bit vector
 * @return d_data 
 */
d_data loadGPUStructures(u_char *text, u_int size, 
                         u_char** patterns, u_int m, u_int nPatt, 
                         bool* bitVector, u_int z,
                         u_long* SGCPFT, u_int nSamP, u_long* PhraFT,
						 u_long* SGCPT, u_int lgPT, u_long* PhraT){

	printf("Loading patterns with size %d\n", nPatt);

	u_char **d_pattern = (u_char **) malloc (nPatt*sizeof(u_char*));
	for (int i = 0; i < nPatt; ++i){
		hipMalloc((void **) &d_pattern[i], (m+1)*sizeof(u_char));
		hipMemcpy(d_pattern[i], patterns[i],(m+1)*sizeof(u_char), hipMemcpyHostToDevice);
	}

	printf("Loading bytes with size %d\n", size);


    int bytes = size*sizeof(u_char);
    u_char* d_text;
	hipMalloc((void **) &d_text, bytes);
	hipMemcpy(d_text, text, bytes, hipMemcpyHostToDevice);

	printf("Loading bit vector with size %d\n", z);


	bool* d_bitVector;
	hipMalloc((void**) &d_bitVector,z*sizeof(bool));
	hipMemcpy(d_bitVector, bitVector, z*sizeof(bool), hipMemcpyHostToDevice);

	printf("Loading sgcpft with size %d\n", nSamP);

	u_long* d_sgcpft;
	hipMalloc((void**) &d_sgcpft, nSamP*sizeof(u_long));
	hipMemcpy(d_sgcpft, SGCPFT, nSamP*sizeof(u_long), hipMemcpyHostToDevice);

	printf("Loading sgcpt with size %d\n", nSamP);

	u_long* d_sgcpt;
	hipMalloc((void**) &d_sgcpt, nSamP*sizeof(u_long));
	hipMemcpy(d_sgcpt, SGCPT, nSamP*sizeof(u_long), hipMemcpyHostToDevice);

	printf("Loading phraFT with size %d\n", z);

	u_long* d_phraft;
	hipMalloc((void**) &d_phraft, z*sizeof(u_long));
	hipMemcpy(d_phraft, PhraFT, z*sizeof(u_long), hipMemcpyHostToDevice);


	u_long lenArray = z*lgPT / (8*sizeof(u_long));
	if ((z*lgPT) % (8*sizeof(u_long)))
		lenArray++;
	u_long* d_phrat;
	printf("Loading phraT with size %d\n", lenArray);
	hipMalloc((void**) &d_phrat, lenArray*sizeof(u_long));
	hipMemcpy(d_phrat, PhraT, lenArray*sizeof(u_long), hipMemcpyHostToDevice);

	u_long* d_occs;
	hipMalloc((void**) &d_occs, size*sizeof(u_long));


	printf("Data structures loaded into the GPU!\n");

	return d_data{
		d_text,
		d_pattern,
		d_bitVector,
		d_sgcpft,
		d_phraft,
		d_sgcpt,
		d_phrat,
		d_occs
	};
}

void prepareOccs(u_long* d_occs, u_long* occs, u_int nFT){
	for(int i = 0; i < nFT; ++i){
		occs[i] = 0;
	}
	printf("occs de 0 %d y nFT %d\n",occs[0], nFT);
	hipMemcpy(d_occs, occs, nFT*sizeof(u_long), hipMemcpyHostToDevice);
}
