#include "hip/hip_runtime.h"
#include "kernel.cuh"

__global__ void kernel_call(unsigned char *text, unsigned char *pattern, int *occs, int size, int m, int *cont){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	bool flag = true;
	int c = 0;
	
	if (tid < size - m){
		for (int i = tid; i < tid+m; ++i){
			if(text[i] != pattern[c]){	
				flag = false;	
			}
			c++;
		}
		if (flag == 1) {occs[tid] = 0; atomicAdd(cont, 1);};
	}
	if (tid == 0){
		printf("pattern: ");
		for (int i = 0; i < m; ++i){
			printf("%c", pattern[i]);
		}
		printf("\n");
	}
}

void calling(unsigned char *text, unsigned char *pattern, int *occs, int size, int m, float &time, long &c){
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	int bytes = size*sizeof(unsigned char);

	//unsigned char *d_text;
	// char	*d_pattern;
	int *d_occs;
	int *d_cont;
	int cont = 0;
	//hipMalloc((void **) &d_text, size*sizeof(unsigned char));
	// hipMalloc((void **) &d_pattern, m*sizeof(char));
	hipMalloc((void **) &d_occs, size*sizeof(int));	
	hipMalloc((void **) &d_cont, sizeof(int));


	//hipMemcpy(d_text, text, bytes, hipMemcpyHostToDevice);
	// hipMemcpy(d_pattern, pattern, m*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(d_occs, occs, size*sizeof(int), hipMemcpyHostToDevice);	
	hipMemcpy(d_cont, &cont, 1*sizeof(int), hipMemcpyHostToDevice);
	hipEventRecord(start,0);

	kernel_call<<<(size+1023)/1024, 1024>>>(text, pattern, d_occs, size, m, d_cont);
	hipMemcpy(occs, d_occs, size*sizeof(int), hipMemcpyDeviceToHost); 
	hipMemcpy(&cont, d_cont, 1*sizeof(int), hipMemcpyDeviceToHost);

	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	time = time/1000.0f;
	
	long acum = 0;
	
	// for(int i = 0; i < size; ++i){
	// 	if(occs[i] == 0){
	// 		c++;
	// 		acum++;
	// 	}
	// }
	c+=cont;
	acum+=cont;
	printf("cont=%d, acum=%d, c=%i\n", cont, acum,c);
	hipFree(d_occs);
	hipFree(d_cont);
	// hipFree(d_pattern);
}


dvc load(unsigned char *text, int size, unsigned char** pattern, int m){
	unsigned char *d_text;
	unsigned char **d_pattern = (unsigned char **) malloc (1000*sizeof(unsigned char*));
	int bytes = size*sizeof(unsigned char);
	hipMalloc((void **) &d_text, size*sizeof(unsigned char));
	
	for (int i = 0; i < 1000; ++i){
		hipMalloc((void **) &d_pattern[i], (m+1)*sizeof(unsigned char));
		
		// hipMemcpy(&d_pattern, &aux, sizeof(unsigned char *), hipMemcpyHostToDevice);
		// hipFree(aux);
	}

	for (int i = 0; i < 1000; ++i){
		hipMemcpy(d_pattern[i], pattern[i],(m+1)*sizeof(unsigned char), hipMemcpyHostToDevice);
	}
	hipMemcpy(d_text, text, bytes, hipMemcpyHostToDevice);

	return dvc{
		d_text,
		d_pattern
	};
}